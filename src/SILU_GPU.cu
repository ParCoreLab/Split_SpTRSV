#include "hip/hip_runtime.h"
// SILU_GPU.cpp
// Contains SILU GPU CUDA kernels and calls
// Author: Najeeb Ahmad
// Created: 24-12-2019

#include "SILU_GPU.h"

int SILU::SyncFreeAnalyzer(device_data_csc *device_csc_matrix, int *inDegree)
{
	int num_threads = 128;
    int num_blocks = ceil ((double)device_csc_matrix->nnz / (double)num_threads);
    hipMemset(inDegree, 0, device_csc_matrix->m * sizeof(int));
       
    if(num_blocks >= 1)
    sptrsv_syncfree_cuda_analyser<<<num_blocks, num_threads>>>
                                (device_csc_matrix->d_cscRowIdx, 
                                device_csc_matrix->m, 
                                device_csc_matrix->nnz, 
                                inDegree);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
        printf("CUDA SyncFree/SLFC Analyzer Error: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    else
        return EXIT_SUCCESS;
}

int SILU::SyncFreeExecutor(device_data_csc *device_csc_matrix, val_type *d_x, val_type *d_b, int *inDegree, int direction)
{
	int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int num_blocks = ceil ((double)device_csc_matrix->m / (double)(num_threads/WARP_SIZE));
    if(num_blocks >= 1)
    sptrsv_syncfree_cuda_executor<<<num_blocks, num_threads >>>
                                (device_csc_matrix->d_cscColPtr, device_csc_matrix->d_cscRowIdx,
                                 device_csc_matrix->d_cscVal, inDegree, 
                                 device_csc_matrix->d_left_sum,
                                 device_csc_matrix->m, direction, d_b, d_x);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
        printf("CUDA SyncFree Executor Error: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    else
        return EXIT_SUCCESS;
}

int SILU::SLFCExecutor(device_data_csc *device_csc_matrix, val_type *d_x,  
                       val_type *diag, ind_type *jlev, int *inDegree)
{
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int num_blocks = ceil ((double)device_csc_matrix->m / (double)(num_threads/WARP_SIZE));

    if(num_blocks >= 1)
    {
        SLFCKernel<<<num_blocks, num_threads>>>
                                (device_csc_matrix->m, d_x, device_csc_matrix->d_cscVal, 
                                 device_csc_matrix->d_cscColPtr, device_csc_matrix->d_cscRowIdx,
                                 diag, inDegree, device_csc_matrix->jlev);
        hipDeviceSynchronize();
    }
    
    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
        printf("CUDA SLFC Executor Error: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    else
        return EXIT_SUCCESS;
}

int SILU::ELMRExecutor(device_data_csr *device_csr_matrix, val_type *d_x, 
                       val_type *d_b, char *ready)
{

    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int num_blocks = ceil ((double)device_csr_matrix->m / (double)(num_threads/WARP_SIZE));


    if(num_blocks >= 1)
    {
        ELMRKernel<<<num_blocks, num_threads>>>
                            (device_csr_matrix->n, d_b, d_x, device_csr_matrix->diag, device_csr_matrix->d_csrVal, 
                             device_csr_matrix->d_csrColIdx, device_csr_matrix->d_csrRowPtr,
                             device_csr_matrix->jlev, ready);
    }    
    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
        printf("CUDA ELMR Executor Error: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    else
    {
        //printf("ELMR Kernel success\n");
        return EXIT_SUCCESS;
    }
    
    //return EXIT_SUCCESS;
}

int SILU::ELMCExecutor(device_data_csc *device_csc_matrix, val_type *d_x, 
                       val_type *d_b, ind_type *count)
{
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int num_blocks = ceil ((double)device_csc_matrix->m / (double)(num_threads/WARP_SIZE));

    if(num_blocks >= 1)
    {
        ELMCKernel<<<num_blocks, num_threads>>>
                    (device_csc_matrix->n, d_x, device_csc_matrix->diag, device_csc_matrix->d_cscVal,
                     device_csc_matrix->d_cscRowIdx, device_csc_matrix->d_cscColPtr,
                     device_csc_matrix->jlev, count);
    }
    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
        printf("CUDA ELMC Executor Error: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    else
    {
        //printf("ELMC Kernel success\n");
        return EXIT_SUCCESS;
    }
}